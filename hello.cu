#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <iostream>

// Для использования просто скомпилируйте программу и выставите в main() входные поля SampleImageF1name[] и SampleImageF2name путь к изображениям одного размера
// Например 
//     char SampleImageF1name[] = "nature.bmp";
//     char SampleImageF2name[] = "dandelion.bmp";
// Или
//     char SampleImageF1name[] = "gradient.bmp";
//     char SampleImageF2name[] = "teapot512.bmp";
// Выходом программы являются два идентичных файла (один сформирован через CUDA, второй через обычные вычисления на CPU)
// А также - главное - разница во времени в выполнении операций на CPU и GPU

using namespace std;

#define BLOCK_SIZE 8

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#pragma pack(push)
#endif
#pragma pack(1)

typedef char int8;
typedef short int16;
typedef int int32;
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint32;

typedef unsigned char byte;

typedef struct {
    uint16 _bm_signature;    //!< File signature, must be "BM"
    uint32 _bm_file_size;    //!< File size
    uint32 _bm_reserved;     //!< Reserved, must be zero
    uint32 _bm_bitmap_data;  //!< Bitmap data
} BMPFileHeader;

typedef struct {
    uint32 _bm_info_header_size;      //!< Info header size, must be 40
    uint32 _bm_image_width;           //!< Image width
    uint32 _bm_image_height;          //!< Image height
    uint16 _bm_num_of_planes;         //!< Amount of image planes, must be 1
    uint16 _bm_color_depth;           //!< Color depth
    uint32 _bm_compressed;            //!< Image compression, must be none
    uint32 _bm_bitmap_size;           //!< Size of bitmap data
    uint32 _bm_hor_resolution;        //!< Horizontal resolution, assumed to be 0
    uint32 _bm_ver_resolution;        //!< Vertical resolution, assumed to be 0
    uint32 _bm_num_colors_used;       //!< Number of colors used, assumed to be 0
    uint32 _bm_num_important_colors;  //!< Number of important colors, assumed to be 0
} BMPInfoHeader;

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#pragma pack(pop)
#else
#pragma pack()
#endif

typedef struct {
    int width;   //!< ROI width
    int height;  //!< ROI height
} ROI;

int clamp_0_255(int x);
byte *MallocPlaneByte(int width, int height, int *pStepBytes);
void FreePlane(void *ptr);
int PreLoadBmp(char *FileName, int *Width, int *Height);
void LoadBmp(char *FileName, int Stride, ROI ImSize, byte *Img);
void DumpBmp(char *FileName, byte *Img, int Stride, ROI ImSize);

__global__ void overlayByteArrays(byte *a, byte *b, byte *c, ROI Size, int threads_count = 0){
    int idx = threadIdx.x;
    for (int i = idx; i < Size.height * Size.width * 3; i += threads_count){
        c[i] = (b[i] + a[i]) / 2;
    }
}

double overlay_images(byte *ImgSrc1, byte *ImgSrc2, byte *ImgDst, int Stride, ROI Size) {
    // create and start timer
    clock_t t1, t2;
    double time_diff;
    t1 = clock();

    // perform overlay
    for (int i = 0; i < Size.height; i++) {
        for (int j = 0; j < Size.width; j++) {
            ImgDst[i * Stride + j * 3] = (ImgSrc1[i * Stride + j * 3] + ImgSrc2[i * Stride + j * 3]) / 2;
            ImgDst[i * Stride + j * 3 + 1] = (ImgSrc1[i * Stride + j * 3 + 1] + ImgSrc2[i * Stride + j * 3 + 1]) / 2;
            ImgDst[i * Stride + j * 3 + 2] = (ImgSrc1[i * Stride + j * 3 + 2] + ImgSrc2[i * Stride + j * 3 + 2]) / 2;
        }
    } 

    // stop and destroy timer
    t2 = clock();
    time_diff = ((double)(t2 - t1)) / CLOCKS_PER_SEC;

    // return time taken by the operation
    return time_diff;
}

double overlay_images_CUDA(byte *ImgSrc1, byte *ImgSrc2, byte *ImgDst, int Stride, ROI Size) {
    byte *da, *db, *dc;
    int size = (((int) ceil(Size.width / 16.0f)) * 16 * 3) * Size.height;

    //move to GPU
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    hipMemcpy(da, ImgSrc1, size, hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(db, ImgSrc2, size, hipMemcpyKind::hipMemcpyHostToDevice);
    
    // create and start timer
    clock_t t1, t2;
    double time_diff;
    t1 = clock();

    // perform overlay
    int threads_count = 1 << 10;
    overlayByteArrays<<<1, threads_count>>>(da, db, dc, Size, threads_count);

    // stop and destroy timer
    t2 = clock();
    time_diff = ((double)(t2 - t1)) / CLOCKS_PER_SEC;

    //get result
    hipMemcpy(ImgDst, dc, size, hipMemcpyKind::hipMemcpyDeviceToHost);

    //Free buffers
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // return time taken by the operation
    return time_diff;
}

int main(){
    char SampleImageF1name[] = "nature.bmp";
    char SampleImageF2name[] = "dandelion.bmp";
    char SampleImageFnameRes[] = "res.bmp";
    char SampleImageFnameResCuda[] = "res_CUDA.bmp";
    char *pSampleImageF1path = SampleImageF1name;
    char *pSampleImageF2path = SampleImageF2name;

    // preload image (acquire dimensions)
    int ImgWidth, ImgHeight;
    int ImgWidth1, ImgHeight1;
    ROI ImgSize;
    int res = PreLoadBmp(pSampleImageF1path, &ImgWidth, &ImgHeight);
    int res2 = PreLoadBmp(pSampleImageF2path, &ImgWidth1, &ImgHeight1);
    ImgSize.width = ImgWidth;
    ImgSize.height = ImgHeight;

    // CONSOLE INFORMATION: saying hello to user
    printf("Loading test images: %s and %s... ", SampleImageF1name, SampleImageF2name);
    if (res) {
        printf("\nError %d: Image file not found or invalid!\n", res);
        exit(EXIT_FAILURE);
        return 1;
    }

    if (ImgHeight != ImgHeight1 || ImgWidth != ImgWidth1){
        printf("\nError: Input image dimensions must be identical!\n");
        exit(EXIT_FAILURE);
        return 1;
    }

    // check image dimensions are multiples of BLOCK_SIZE
    if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0) {
        printf("\nError: Input image dimensions must be multiples of 8!\n");
        exit(EXIT_FAILURE);
        return 1;
    }

    printf("[%d x %d]... \n", ImgWidth, ImgHeight);

    // allocate image buffers
    int ImgStride;
    byte *ImgSrc1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    byte *ImgSrc2 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    byte *ImgDst = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    byte *ImgDstCUDA = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);

    // load images
    LoadBmp(pSampleImageF1path, ImgStride, ImgSize, ImgSrc1);
    LoadBmp(pSampleImageF2path, ImgStride, ImgSize, ImgSrc2);

    //Running CPU version
    printf("Success\nRunning CPU version... \n");
    double TimeCPU = overlay_images(ImgSrc1, ImgSrc2, ImgDst, ImgStride, ImgSize);

    //Running GPU version
    printf("Success\nRunning GPU version... \n");
    double TimeGPU = overlay_images_CUDA(ImgSrc1, ImgSrc2, ImgDstCUDA, ImgStride, ImgSize);

    printf("Success\nDumping result to %s... \n", SampleImageFnameRes);
    DumpBmp(SampleImageFnameRes, ImgDst, ImgStride, ImgSize);

    printf("Success\nDumping result to %s... \n", SampleImageFnameResCuda);
    DumpBmp(SampleImageFnameResCuda, ImgDstCUDA, ImgStride, ImgSize);

    //Finalize
    FreePlane(ImgSrc1);
    FreePlane(ImgSrc2);
    FreePlane(ImgDst);
    FreePlane(ImgDstCUDA);

    printf("Processing time (CPU)    : %f ms \n", TimeCPU);
    printf("Processing time (GPU)    : %f ms \n", TimeGPU);

    return 0;
}

int clamp_0_255(int x) {
    return (x < 0) ? 0 : ((x > 255) ? 255 : x); 
}

byte *MallocPlaneByte(int width, int height, int *pStepBytes) {
    byte *ptr;
    *pStepBytes = ((int) ceil(width / 16.0f)) * 16 * 3;
    ptr = (byte *) malloc(*pStepBytes * height);
    return ptr;
}

void FreePlane(void *ptr) {
  if (ptr) {
    free(ptr);
  }
}

int PreLoadBmp(char *FileName, int *Width, int *Height) {
    BMPFileHeader FileHeader;
    BMPInfoHeader InfoHeader;
    FILE *fh;

    if (!(fh = fopen(FileName, "rb"))) {
        return 1;  // invalid filename
    }

    fread(&FileHeader, sizeof(BMPFileHeader), 1, fh);

    if (FileHeader._bm_signature != 0x4D42) {
        return 2;  // invalid file format
    }

    fread(&InfoHeader, sizeof(BMPInfoHeader), 1, fh);

    if (InfoHeader._bm_color_depth != 24) {
        printf("depth is %d\n", InfoHeader._bm_color_depth);
        return 3;  // invalid color depth
    }

    if (InfoHeader._bm_compressed) {
        printf("compression is %d\n", InfoHeader._bm_compressed);
        return 4;  // invalid compression property
    }

    *Width = InfoHeader._bm_image_width;
    *Height = InfoHeader._bm_image_height;

    fclose(fh);

    return 0;
}

void LoadBmp(char *FileName, int Stride, ROI ImSize, byte *Img) {
    BMPFileHeader FileHeader;
    BMPInfoHeader InfoHeader;
    FILE *fh;
    fh = fopen(FileName, "rb");

    fread(&FileHeader, sizeof(BMPFileHeader), 1, fh);
    fread(&InfoHeader, sizeof(BMPInfoHeader), 1, fh);

    for (int i = ImSize.height - 1; i >= 0; i--) {
        for (int j = 0; j < ImSize.width; j++) {
            int r = 0, g = 0, b = 0;
            fread(&b, 1, 1, fh);
            fread(&g, 1, 1, fh);
            fread(&r, 1, 1, fh);
            Img[i * Stride + j * 3] = b;
            Img[i * Stride + j * 3 + 1] = g;
            Img[i * Stride + j * 3 + 2] = r;
        }
    }

    fclose(fh);
    return;
}

void DumpBmp(char *FileName, byte *Img, int Stride, ROI ImSize) {
    FILE *fp = NULL;
    fp = fopen(FileName, "wb");

    if (fp == NULL) {
        return;
    }

    BMPFileHeader FileHeader;
    BMPInfoHeader InfoHeader;

    // init headers
    FileHeader._bm_signature = 0x4D42;
    FileHeader._bm_file_size = 54 + 3 * ImSize.width * ImSize.height;
    FileHeader._bm_reserved = 0;
    FileHeader._bm_bitmap_data = 0x36;
    InfoHeader._bm_bitmap_size = 0;
    InfoHeader._bm_color_depth = 24;
    InfoHeader._bm_compressed = 0;
    InfoHeader._bm_hor_resolution = 0;
    InfoHeader._bm_image_height = ImSize.height;
    InfoHeader._bm_image_width = ImSize.width;
    InfoHeader._bm_info_header_size = 40;
    InfoHeader._bm_num_colors_used = 0;
    InfoHeader._bm_num_important_colors = 0;
    InfoHeader._bm_num_of_planes = 1;
    InfoHeader._bm_ver_resolution = 0;

    fwrite(&FileHeader, sizeof(BMPFileHeader), 1, fp);
    fwrite(&InfoHeader, sizeof(BMPInfoHeader), 1, fp);

    for (int i = ImSize.height - 1; i >= 0; i--) {
        for (int j = 0; j < ImSize.width; j++) {
        fwrite(&(Img[i * Stride + j * 3]), 1, 1, fp);
        fwrite(&(Img[i * Stride + j * 3 + 1]), 1, 1, fp);
        fwrite(&(Img[i * Stride + j * 3 + 2]), 1, 1, fp);
        }
    }

    fclose(fp);
}
